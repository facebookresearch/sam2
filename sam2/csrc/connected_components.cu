#include "hip/hip_runtime.h"
// Copyright (c) Meta Platforms, Inc. and affiliates.
// All rights reserved.

// This source code is licensed under the license found in the
// LICENSE file in the root directory of this source tree.

// adapted from https://github.com/zsef123/Connected_components_PyTorch
// with license found in the LICENSE_cctorch file in the root directory.
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <torch/script.h>
#include <vector>

// 2d
#define BLOCK_ROWS 16
#define BLOCK_COLS 16

namespace cc2d {

template <typename T>
__device__ __forceinline__ unsigned char hasBit(T bitmap, unsigned char pos) {
  return (bitmap >> pos) & 1;
}

__device__ int32_t find(const int32_t* s_buf, int32_t n) {
  while (s_buf[n] != n)
    n = s_buf[n];
  return n;
}

__device__ int32_t find_n_compress(int32_t* s_buf, int32_t n) {
  const int32_t id = n;
  while (s_buf[n] != n) {
    n = s_buf[n];
    s_buf[id] = n;
  }
  return n;
}

__device__ void union_(int32_t* s_buf, int32_t a, int32_t b) {
  bool done;
  do {
    a = find(s_buf, a);
    b = find(s_buf, b);

    if (a < b) {
      int32_t old = atomicMin(s_buf + b, a);
      done = (old == b);
      b = old;
    } else if (b < a) {
      int32_t old = atomicMin(s_buf + a, b);
      done = (old == a);
      a = old;
    } else
      done = true;

  } while (!done);
}

__global__ void
init_labeling(int32_t* label, const uint32_t W, const uint32_t H) {
  const uint32_t row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
  const uint32_t col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
  const uint32_t idx = row * W + col;

  if (row < H && col < W)
    label[idx] = idx;
}

__global__ void
merge(uint8_t* img, int32_t* label, const uint32_t W, const uint32_t H) {
  const uint32_t row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
  const uint32_t col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
  const uint32_t idx = row * W + col;

  if (row >= H || col >= W)
    return;

  uint32_t P = 0;

  if (img[idx])
    P |= 0x777;
  if (row + 1 < H && img[idx + W])
    P |= 0x777 << 4;
  if (col + 1 < W && img[idx + 1])
    P |= 0x777 << 1;

  if (col == 0)
    P &= 0xEEEE;
  if (col + 1 >= W)
    P &= 0x3333;
  else if (col + 2 >= W)
    P &= 0x7777;

  if (row == 0)
    P &= 0xFFF0;
  if (row + 1 >= H)
    P &= 0xFF;

  if (P > 0) {
    // If need check about top-left pixel(if flag the first bit) and hit the
    // top-left pixel
    if (hasBit(P, 0) && img[idx - W - 1]) {
      union_(label, idx, idx - 2 * W - 2); // top left block
    }

    if ((hasBit(P, 1) && img[idx - W]) || (hasBit(P, 2) && img[idx - W + 1]))
      union_(label, idx, idx - 2 * W); // top bottom block

    if (hasBit(P, 3) && img[idx + 2 - W])
      union_(label, idx, idx - 2 * W + 2); // top right block

    if ((hasBit(P, 4) && img[idx - 1]) || (hasBit(P, 8) && img[idx + W - 1]))
      union_(label, idx, idx - 2); // just left block
  }
}

__global__ void compression(int32_t* label, const int32_t W, const int32_t H) {
  const uint32_t row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
  const uint32_t col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
  const uint32_t idx = row * W + col;

  if (row < H && col < W)
    find_n_compress(label, idx);
}

__global__ void final_labeling(
    const uint8_t* img,
    int32_t* label,
    const int32_t W,
    const int32_t H) {
  const uint32_t row = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
  const uint32_t col = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
  const uint32_t idx = row * W + col;

  if (row >= H || col >= W)
    return;

  int32_t y = label[idx] + 1;

  if (img[idx])
    label[idx] = y;
  else
    label[idx] = 0;

  if (col + 1 < W) {
    if (img[idx + 1])
      label[idx + 1] = y;
    else
      label[idx + 1] = 0;

    if (row + 1 < H) {
      if (img[idx + W + 1])
        label[idx + W + 1] = y;
      else
        label[idx + W + 1] = 0;
    }
  }

  if (row + 1 < H) {
    if (img[idx + W])
      label[idx + W] = y;
    else
      label[idx + W] = 0;
  }
}

__global__ void init_counting(
    const int32_t* label,
    int32_t* count_init,
    const int32_t W,
    const int32_t H) {
  const uint32_t row = (blockIdx.y * blockDim.y + threadIdx.y);
  const uint32_t col = (blockIdx.x * blockDim.x + threadIdx.x);
  const uint32_t idx = row * W + col;

  if (row >= H || col >= W)
    return;

  int32_t y = label[idx];
  if (y > 0) {
    int32_t count_idx = y - 1;
    atomicAdd(count_init + count_idx, 1);
  }
}

__global__ void final_counting(
    const int32_t* label,
    const int32_t* count_init,
    int32_t* count_final,
    const int32_t W,
    const int32_t H) {
  const uint32_t row = (blockIdx.y * blockDim.y + threadIdx.y);
  const uint32_t col = (blockIdx.x * blockDim.x + threadIdx.x);
  const uint32_t idx = row * W + col;

  if (row >= H || col >= W)
    return;

  int32_t y = label[idx];
  if (y > 0) {
    int32_t count_idx = y - 1;
    count_final[idx] = count_init[count_idx];
  } else {
    count_final[idx] = 0;
  }
}

} // namespace cc2d

std::vector<torch::Tensor> get_connected_components(
    const torch::Tensor& inputs) {
  AT_ASSERTM(inputs.is_cuda(), "inputs must be a CUDA tensor");
  AT_ASSERTM(inputs.ndimension() == 4, "inputs must be [N, 1, H, W] shape");
  AT_ASSERTM(
      inputs.scalar_type() == torch::kUInt8, "inputs must be a uint8 type");

  const uint32_t N = inputs.size(0);
  const uint32_t C = inputs.size(1);
  const uint32_t H = inputs.size(2);
  const uint32_t W = inputs.size(3);

  AT_ASSERTM(C == 1, "inputs must be [N, 1, H, W] shape");
  AT_ASSERTM((H % 2) == 0, "height must be an even number");
  AT_ASSERTM((W % 2) == 0, "width must be an even number");

  // label must be uint32_t
  auto label_options =
      torch::TensorOptions().dtype(torch::kInt32).device(inputs.device());
  torch::Tensor labels = torch::zeros({N, C, H, W}, label_options);
  torch::Tensor counts_init = torch::zeros({N, C, H, W}, label_options);
  torch::Tensor counts_final = torch::zeros({N, C, H, W}, label_options);

  dim3 grid = dim3(
      ((W + 1) / 2 + BLOCK_COLS - 1) / BLOCK_COLS,
      ((H + 1) / 2 + BLOCK_ROWS - 1) / BLOCK_ROWS);
  dim3 block = dim3(BLOCK_COLS, BLOCK_ROWS);
  dim3 grid_count =
      dim3((W + BLOCK_COLS) / BLOCK_COLS, (H + BLOCK_ROWS) / BLOCK_ROWS);
  dim3 block_count = dim3(BLOCK_COLS, BLOCK_ROWS);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  for (int n = 0; n < N; n++) {
    uint32_t offset = n * H * W;

    cc2d::init_labeling<<<grid, block, 0, stream>>>(
        labels.data_ptr<int32_t>() + offset, W, H);
    cc2d::merge<<<grid, block, 0, stream>>>(
        inputs.data_ptr<uint8_t>() + offset,
        labels.data_ptr<int32_t>() + offset,
        W,
        H);
    cc2d::compression<<<grid, block, 0, stream>>>(
        labels.data_ptr<int32_t>() + offset, W, H);
    cc2d::final_labeling<<<grid, block, 0, stream>>>(
        inputs.data_ptr<uint8_t>() + offset,
        labels.data_ptr<int32_t>() + offset,
        W,
        H);

    // get the counting of each pixel
    cc2d::init_counting<<<grid_count, block_count, 0, stream>>>(
        labels.data_ptr<int32_t>() + offset,
        counts_init.data_ptr<int32_t>() + offset,
        W,
        H);
    cc2d::final_counting<<<grid_count, block_count, 0, stream>>>(
        labels.data_ptr<int32_t>() + offset,
        counts_init.data_ptr<int32_t>() + offset,
        counts_final.data_ptr<int32_t>() + offset,
        W,
        H);
  }

  // returned values are [labels, counts]
  std::vector<torch::Tensor> outputs;
  outputs.push_back(labels);
  outputs.push_back(counts_final);
  return outputs;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def(
      "get_connected_components",
      &get_connected_components,
      "get_connected_components");
}
